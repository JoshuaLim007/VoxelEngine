#include "hip/hip_runtime.h"
// DDARaytracer.cpp : This file contains the 'main' function. Program execution begins and ends there.
//

#include <iostream>
#include "../SDLRenderer/SDLRenderer.h"
#include "../GPUDDA/DDA.cuh"
#include "../GPUDDA/Raytracer.cuh"
#include "../GPUDDA/VoxelWorldBuilder.cuh"
#include <fstream>
#include <sstream>

using namespace GPUDDA::Graphics;
using namespace GPUDDA;

VoxelBuffer<3> CreateVoxels(uint3 size) {
	VoxelBuffer<3> voxels;
	voxels.dimensions[0] = size.x;
	voxels.dimensions[1] = size.y;
	voxels.dimensions[2] = size.z;
	size_t buffer_size = static_cast<size_t>(size.x) * size.y * size.z;
	voxels.grid = BitArray(buffer_size);

	BitArray temp = BitArray(buffer_size, true);
	auto threads = dim3(8, 8, 8);
	auto scaled_size = make_uint3(size.x, size.y, size.z);
	auto dim = dim3(
		(scaled_size.x / 8 + threads.x - 1) / threads.x, 
		(scaled_size.y + threads.y - 1) / threads.y,
		(scaled_size.z + threads.z - 1) / threads.z);

	PopulateVoxels << <dim, threads >> > (temp, scaled_size);
	CUDA_SAFE_CALL(hipDeviceSynchronize());
	hipMemcpy(voxels.grid.raw(), temp.raw(), temp.byte_size(), hipMemcpyDeviceToHost);
	hipFree(temp.raw());

	return voxels;
}
int main()
{
	int factor = 32;
	auto t0 = std::chrono::high_resolution_clock::now();
	auto buffer = CreateVoxels(make_uint3(2048 * 4, 512, 2048 * 4));
	auto t1 = std::chrono::high_resolution_clock::now();
	auto td = std::chrono::duration_cast<std::chrono::milliseconds>(t1 - t0).count();
	std::cout << "Voxel generation time: " << td << "ms" << std::endl;

	auto t2 = std::chrono::high_resolution_clock::now();
	auto buffers = createBuffersFromVoxels(buffer, factor);
	auto t3 = std::chrono::high_resolution_clock::now();
	auto td2 = std::chrono::duration_cast<std::chrono::milliseconds>(t3 - t2).count();
	std::cout << "Buffer generation time: " << td2 << "ms" << std::endl;

	delete[] buffer.grid.raw();
	Renderer renderer("SDL Window");
	if (!renderer.init(1920, 1080)) {
		return 1;
	}
	VoxelRaytracer3D* raytracer = new GPUDDA::VoxelRaytracer3D(1);
	auto low_res_buffer = std::get<0>(buffers);
	auto low_res_grid_data = std::get<1>(buffers);
	auto bounds = std::get<2>(buffers);
	auto count = low_res_buffer.dimensions[0] * low_res_buffer.dimensions[1] * low_res_buffer.dimensions[2];
	raytracer->UploadVoxelBuffer(low_res_buffer);
	raytracer->UploadVoxelBufferDatas(low_res_grid_data, count);
	raytracer->UploadVoxelBufferDataBounds(bounds, count);
	raytracer->SetFactor(factor);

	void* d_pixels;
	float3 cam_pos = { 0, 0, 0 };
	float3 cam_up = { 0, 1, 0 };
	float3 cam_right = { 1, 0, 0 };
	float3 cam_forward = { 0, 0, 1 };
	float3 cam_eular = { 0, 0, 0 };

	Graphics::Environment env;
	env.LightDirection = { 0, 1, 0};
	env.LightDirection = normalize(env.LightDirection);
	env.LightColor = { 10, 10, 10 };
	env.AmbientColor = { 0.5f, 0.5f, 0.5f };
	SetEnvironment(env);

	hipMalloc(&d_pixels, 1920 * 1080 * sizeof(PixelData));
	hipMemset(d_pixels, 255, 1920 * 1080 * sizeof(PixelData));
	bool clicking = false;
	renderer.AddRenderEventCallback([&](const CallbackData& data) {
		SDL_Event e;
		while (SDL_PollEvent(&e)) {
			if (e.type == SDL_QUIT) {
				data.renderer->close();
			}
			if (e.type == SDL_MOUSEBUTTONDOWN) {
				if (e.button.button == SDL_BUTTON_LEFT) {
					clicking = true;
				}
			}
			if (e.type == SDL_MOUSEBUTTONUP) {
				if (e.button.button == SDL_BUTTON_LEFT) {
					clicking = false;
				}
			}
		}

		float cam_speed = 0.05;
		const Uint8* currentKeyStates = SDL_GetKeyboardState(NULL);
		//shift
		if (currentKeyStates[SDL_SCANCODE_LSHIFT]) {
			cam_speed *= 100;
		}
		if (currentKeyStates[SDL_SCANCODE_W]) {
			cam_pos += cam_forward * cam_speed;
		}
		if (currentKeyStates[SDL_SCANCODE_S]) {
			cam_pos -= cam_forward * cam_speed;
		}
		if (currentKeyStates[SDL_SCANCODE_A]) {
			cam_pos -= cam_right * cam_speed;
		}
		if (currentKeyStates[SDL_SCANCODE_D]) {
			cam_pos += cam_right * cam_speed;
		}
		if (currentKeyStates[SDL_SCANCODE_Q]) {
			cam_pos -= cam_up * cam_speed;
		}
		if (currentKeyStates[SDL_SCANCODE_E]) {
			cam_pos += cam_up * cam_speed;
		}

		if (currentKeyStates[SDL_SCANCODE_R]) {
			//read from text file 
			std::fstream file("C:\\Users\\joshu\\Desktop\\camData.txt", std::ios::in);
			//first line contains rotation
			std::string line;
			std::getline(file, line);
			std::istringstream iss(line);
			std::string token;
			std::getline(iss, token, ',');
			cam_eular.x = std::stof(token);
			std::getline(iss, token, ',');
			cam_eular.y = std::stof(token);
			std::getline(iss, token, ',');
			cam_eular.z = std::stof(token);
			//second line contains position
			std::getline(file, line);
			std::istringstream iss2(line);
			std::getline(iss2, token, ',');
			cam_pos.x = std::stof(token);
			std::getline(iss2, token, ',');
			cam_pos.y = std::stof(token);
			std::getline(iss2, token, ',');
			cam_pos.z = std::stof(token);
			file.close();

			//cam_pos = { 138.252, 101.042, 503.197 };
			//cam_eular = { -0.522997, 0.512, 0 };
		}

		std::cout << "Cam pos: " << cam_pos.x << ", " << cam_pos.y << ", " << cam_pos.z << std::endl;
		std::cout << "Cam eular: " << cam_eular.x << ", " << cam_eular.y << ", " << cam_eular.z << std::endl;


		static int last_x = 0, last_y = 0;
		int x, y;
		SDL_GetMouseState(&x, &y);
		if (clicking) {
			//mouse movement
			int dx = x - last_x;
			int dy = y - last_y;
			cam_eular.x += dy * 0.001f;
			cam_eular.y += dx * 0.001f;
		}
		last_x = x;
		last_y = y;

		std::cout << "Cam Forward: " << cam_forward.x << ", " << cam_forward.y << ", " << cam_forward.z << std::endl;

		getDirections(cam_eular, &cam_forward, &cam_up, &cam_right);
		RaytraceScreen(raytracer, 1920, 1080, d_pixels, cam_pos, cam_forward, cam_up, cam_right);
		hipMemcpy(data.pixels, d_pixels, 1920 * 1080 * sizeof(PixelData), hipMemcpyDeviceToHost);
		});

	bool running = true;
	while (running) {
		auto t0 = std::chrono::high_resolution_clock::now();
		running = renderer.render();
		auto t1 = std::chrono::high_resolution_clock::now();
		auto td = std::chrono::duration_cast<std::chrono::milliseconds>(t1 - t0).count();
		printf("Frame time: %dms\n", td);
		auto fps = 1000.0f / td;
		std::cout << "FPS: " << fps << std::endl;
	}
	hipFree(d_pixels);
	delete raytracer;
}
