#include "hip/hip_runtime.h"
#include "Raytracer.cuh"
#include "cuda_noise.cuh"
using namespace GPUDDA;

struct RenderParams {
	uint2 Resolution;
	size_t FrameNumber;
	float Fov;
	float2 OrthoSize;
	__device__ __host__ RenderParams(uint2 r, size_t n, float fov, float2 size) {
		Resolution = r;
		FrameNumber = n;
		Fov = fov;
		OrthoSize = size;
	}
	__device__ __host__ RenderParams() {}
};
__device__ RenderParams d_params;
RenderParams h_params = RenderParams(make_uint2(0,0), 0, 90, make_float2(10,10));

__host__ __device__ void Graphics::getDirections(float3 eularAngles, float3* forwad, float3* up, float3* right)
{
	float3 fwd = make_float3(0, 0, 0);
	float3 upVec = make_float3(0, 0, 0);
	float3 rgt = make_float3(0, 0, 0);
	fwd.x = cos(eularAngles.x) * sin(eularAngles.y);
	fwd.y = -sin(eularAngles.x);
	fwd.z = cos(eularAngles.x) * cos(eularAngles.y);
	rgt.x = cos(eularAngles.y);
	rgt.y = 0;
	rgt.z = -sin(eularAngles.y);
	upVec = cross(fwd, rgt);
	*forwad = fwd * -1;
	*up = upVec * -1;
	*right = rgt;
}

__device__ float3 getRayDirection(float3 fwd, float3 up, float3 right, uint2 screen_dim, float3 uv, float FOV) {
	float aspectRatio = (float)screen_dim.x / (float)screen_dim.y;
	uv.x = uv.x * 2 - 1;
	uv.y = uv.y * 2 - 1;
	uv.z = 1;
	float fov = FOV * 3.1415 / 180.0;
	float scale_x = tanf(fov / 2.0f) * aspectRatio;
	float scale_y = tanf(fov / 2.0f);
	float3 ray_dir{};
	ray_dir.x = fwd.x + uv.x * scale_x * right.x + uv.y * scale_y * up.x;
	ray_dir.y = fwd.y + uv.x * scale_x * right.y + uv.y * scale_y * up.y;
	ray_dir.z = fwd.z + uv.x * scale_x * right.z + uv.y * scale_y * up.z;
	ray_dir = normalize(ray_dir);
	return ray_dir;
}

__device__ void getRayDirectionOrtho(
	float3 fwd, 
	float3 up, 
	float3 right, 
	float2 uv, 
	float2 screen_size,
	float3 origin,
	float3& out_rayDir,
	float3& out_rayOrigin) {

	float ratio = static_cast<float>(d_params.Resolution.x) / d_params.Resolution.y;
	out_rayDir = fwd;
	out_rayOrigin = origin;
	out_rayOrigin += right * (uv.x * 2 - 1) * screen_size.x * ratio;
	out_rayOrigin += up * (uv.y * 2 - 1) * screen_size.y;
}

template<typename T>
__device__ void setPixelColor(void* screen_texture, uint32_t screen_width, uint32_t screen_height, int x, int y, float3 color) {
	T* pixels = (T*)screen_texture;
	if (x < screen_width && y < screen_height) {
		T* pixel = &pixels[y * screen_width + x];
		color.x = fminf(fmaxf(color.x, 0), 1);
		color.y = fminf(fmaxf(color.y, 0), 1);
		color.z = fminf(fmaxf(color.z, 0), 1);

		pixel->r = color.x * 255;
		pixel->g = color.y * 255;
		pixel->b = color.z * 255;
		pixel->a = 255;
	}
}

__device__ Graphics::Environment g_env;
__device__ float3 calculateColor(float3 camPos, float3 normal, float3 position,
	VoxelBuffer<3>* chunks,
	VoxelBuffer<3>* chunksData,
	Bounds<float3>* chunkBoundingBoxes,
	int factor,
	int& out_steps) {
	out_steps = 0;

	//shadow
	float3 shadowRay = normalize(g_env.LightDirection);
	float3 shadowPos = position + g_env.LightDirection * 0.01f;
	float3 shadowNormal;
	int steps;
	bool hit = raytrace(MAX_STEPS, shadowPos, shadowRay, chunks[0], chunksData, chunkBoundingBoxes, factor, steps, shadowNormal, shadowPos);
	out_steps += steps;

	float lDot = fmaxf(dot(normal, g_env.LightDirection), 0) * (hit ? 0 : 1);
	float3 diffuse = lDot * g_env.LightColor;
	float3 ambient = g_env.AmbientColor * lerp(0.25,1.0, dot(normal, make_float3(0, 1, 0)) * 0.5 + 0.5 );
	float3 color = diffuse + ambient;

	//specular
	if (!hit) {
		float3 viewDir = normalize(position - camPos);
		float3 reflectDir = reflect(g_env.LightDirection, normal);
		float spec = powf(fmaxf(dot(viewDir, reflectDir), 0), 32);
		color.x += spec * g_env.LightColor.x;
		color.y += spec * g_env.LightColor.y;
		color.z += spec * g_env.LightColor.z;
	}

	//Ambient Occlusion
	if (lDot == 0) {
		constexpr int samples = 8;
		int x = blockIdx.x * blockDim.x + threadIdx.x;
		int y = blockIdx.y * blockDim.y + threadIdx.y;
		int seed = y * 1920 + x;
		float occlusion = 0.0f;
		for (int i = 0; i < samples; i++) {
			int si = seed + i * 1000 + (d_params.FrameNumber + 1) * 1000;
			float3 sampleDir = make_float3(
				cudaNoise::randomFloat(si) * 2 - 1,
				cudaNoise::randomFloat(si * 10) * 2 - 1,
				cudaNoise::randomFloat(si * 100) * 2 - 1);
			sampleDir = normalize(sampleDir);
			if (dot(sampleDir, normal) < 0) {
				sampleDir = reflect(sampleDir, normal);
			}

			float3 samplePos = position + sampleDir * 0.01f;
			float3 sampleNormal;
			bool hit = raytrace(8, samplePos, sampleDir, chunks[0], chunksData, chunkBoundingBoxes, factor, steps, sampleNormal, samplePos);
			if (hit) {
				float dist = length(samplePos - position);
				float occlusion = 1 - fminf(1 / (dist * 10.0f), 1.0f);
				occlusion += occlusion;
			}
			else {
				occlusion += 1.0f;
			}
		}
		occlusion /= samples;
		color *= occlusion;
	}

	return color;
}

__device__ float3 Tonemap(float3 color) {
	float3 tonemappedColor = color / (color + make_float3(1.0f));
	tonemappedColor.x = fminf(fmaxf(tonemappedColor.x, 0), 1);
	tonemappedColor.y = fminf(fmaxf(tonemappedColor.y, 0), 1);
	tonemappedColor.z = fminf(fmaxf(tonemappedColor.z, 0), 1);
	return tonemappedColor;
}

__global__ void screenDispatch(
	float3 origin,
	float3 camera_fwd,
	float3 camera_up,
	float3 camera_right,
	uint32_t screen_width,
	uint32_t screen_height,
	void* screen_texture,

	VoxelBuffer<3>* chunks,
	VoxelBuffer<3>* chunksData,
	Bounds<float3>* chunkBoundingBoxes,
	int factor) {

	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	if (x < screen_width && y < screen_height) {
		float2 uv = make_float2(x / (float)screen_width, y / (float)screen_height);
#ifdef ORTHO
		float3 ray_dir;
		getRayDirectionOrtho(camera_fwd, camera_up, camera_right, uv, d_params.OrthoSize, origin, ray_dir, origin);
#else
		auto ray_dir = getRayDirection(camera_fwd, camera_up, camera_right, make_uint2(screen_width, screen_height), make_float3(uv.x, uv.y, 0), d_params.Fov);
#endif
		int steps;
		float3 normal;
		float3 hitPos;
		bool hit = raytrace(MAX_STEPS, origin, ray_dir, chunks[0], chunksData, chunkBoundingBoxes, factor, steps, normal, hitPos);
		normal = -normal;
		if (hit) {
#ifdef DEBUG_VIEW
			float dist = length(hitPos - origin);
			hitPos.x = (hitPos.x) / 128.0f;
			hitPos.y = (hitPos.y) / 128.0f;
			hitPos.z = (hitPos.z) / 128.0f;
			hitPos.x = fmodf(hitPos.x, 1.0f + FLT_EPS_DDA);
			hitPos.y = fmodf(hitPos.y, 1.0f + FLT_EPS_DDA);
			hitPos.z = fmodf(hitPos.z, 1.0f + FLT_EPS_DDA);

			////top left
			//if (x < screen_width >> 1 && y < screen_height >> 1) {
			//	setPixelColor<Graphics::BGRA8888>(screen_texture, screen_width, screen_height, x, y, make_float3(normal.x, normal.y, normal.z));
			//}
			////top right
			//else if(x > screen_width >> 1 && y < screen_height >> 1){
			//	setPixelColor<Graphics::BGRA8888>(screen_texture, screen_width, screen_height, x, y, make_float3(hitPos.x, hitPos.y, hitPos.z));
			//}
			////bottom left
			//else if (x < screen_width >> 1) {
			//	//nothing
			//}
			////bottom right
			//else {
			//	setPixelColor<Graphics::BGRA8888>(screen_texture, screen_width, screen_height, x, y, make_float3(dist * 0.01f, 0, 0));
			//}

			setPixelColor<Graphics::BGRA8888>(screen_texture, screen_width, screen_height, x, y, make_float3(fmodf(dist, 1.0f),0,0));

#else
			int color_steps = 0;
			float3 color = calculateColor(origin, normal, hitPos, chunks, chunksData, chunkBoundingBoxes, factor, color_steps);
			color = Tonemap(color);
			steps += color_steps;
			setPixelColor<Graphics::BGRA8888>(screen_texture, screen_width, screen_height, x, y, make_float3(color.x, color.y, color.z));
#endif
		}
		else {
			setPixelColor<Graphics::BGRA8888>(screen_texture, screen_width, screen_height, x, y, make_float3(ray_dir.x, ray_dir.y, ray_dir.z));
		}
#ifdef DEBUG_VIEW
		//if (x < screen_width >> 1 && y > screen_height >> 1) {
		//	setPixelColor<Graphics::BGRA8888>(screen_texture, screen_width, screen_height, x, y, make_float3(steps / 256.0f, 0, 0));
		//}
#endif
	}
}

void Graphics::SetEnvironment(const Environment& env_v) {
	void* d_env;
	hipGetSymbolAddress(&d_env, HIP_SYMBOL(g_env));
	auto err = hipGetLastError();
	if (err != hipSuccess) {
		std::cout << "Error: " << hipGetErrorString(err) << std::endl;
	}
	hipMemcpy(d_env, &env_v, sizeof(Environment), hipMemcpyHostToDevice);
	err = hipGetLastError();
	if (err != hipSuccess) {
		std::cout << "Error: " << hipGetErrorString(err) << std::endl;
	}
}

void Graphics::SetFOV(float fov) {
	h_params.Fov = fov;
}

void Graphics::SetOrthoWindowSize(float2 size) {
	h_params.OrthoSize = size;
}

void Graphics::RaytraceScreen(
	VoxelRaytracer3D* rt,
	uint32_t screen_width,
	uint32_t screen_height,
	void* d_screen_texture,
	float3 origin,
	float3 camera_fwd,
	float3 camera_up,
	float3 camera_right) {

	dim3 blockSize(8, 8, 1);
	dim3 numBlocks((screen_width + blockSize.x - 1) / blockSize.x, (screen_height + blockSize.y - 1) / blockSize.y, 1);

	auto buffer = rt->GetVoxelBuffer();
	auto bufferDataBounds = rt->GetVoxelBufferDataBounds();
	auto bufferData = rt->GetVoxelBufferDatas();
	auto factor = rt->GetFactor();
	
	h_params.Resolution = make_uint2(screen_width, screen_height);
	hipMemcpyToSymbol(HIP_SYMBOL(d_params), &h_params, sizeof(RenderParams));
	h_params.FrameNumber++;

	screenDispatch << < numBlocks, blockSize >> > (
		origin, camera_fwd, camera_up, camera_right,
		screen_width, screen_height, d_screen_texture,
		buffer, bufferData, bufferDataBounds, factor);

	CUDA_SAFE_CALL(hipDeviceSynchronize());
}