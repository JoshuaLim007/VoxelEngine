#include "hip/hip_runtime.h"
#include "DDA.cuh"

#include <hip/hip_runtime.h>
#include <>
#include <vector>
#include <chrono>
#include <iostream>
#include "hip/hip_vector_types.h"
#include <device_atomic_functions.h>

namespace GPUDDA {
    __device__ __host__ BitRef::operator bool() const {
        return (*byte >> index) & 1;
    }

    __host__ __device__ BitRef& BitRef::operator=(bool value) {
        if (value)
            *byte |= (1 << index);  // Set bit
        else
            *byte &= ~(1 << index); // Clear bit
        return *this;
    }
    __device__ __host__ BitArray::BitArray() : size(0), data(nullptr) {}
    __device__ __host__ BitArray::BitArray(size_t num_bits) : size(num_bits) {
        data = new uint8_t[(size + 7) / 8]();
    }
    __device__ __host__ BitArray::BitArray(const BitArray& other, bool isGPU) : size(other.size) {
        if (isGPU) {
			hipMalloc((void**)&data, (size + 7) / 8);
			hipMemcpy(data, other.data, (size + 7) / 8, hipMemcpyHostToDevice);
            return;
        }
        data = new uint8_t[(size + 7) / 8];
		std::copy(other.data, other.data + (size + 7) / 8, data);
	}
    __device__ __host__ BitArray::BitArray(size_t num_bits, bool isGPU) : size(num_bits) {
        if (isGPU) {
            hipMalloc((void**)&data, (size + 7) / 8);
            return;
        }
        data = new uint8_t[(size + 7) / 8];
    }
    __device__ __host__ void BitArray::Copy(const BitArray& other, bool isGPU) {
        if (isGPU) {
            hipMemcpy(data, other.data, (size + 7) / 8, hipMemcpyHostToDevice);
            return;
        }
        std::copy(other.data, other.data + (size + 7) / 8, data);
    }
    __device__ __host__ void BitArray::AsyncCopy(const BitArray& other, hipStream_t stream) {
        hipMemcpyAsync(data, other.data, (size + 7) / 8, hipMemcpyHostToDevice, stream);
        return;
    }
    __device__ __host__ bool BitArray::operator[](size_t index) const {
		if (index >= size) {
			return false; // Out of bounds
		}
        return (data[index / 8] >> (index % 8)) & 1;
    }

    __device__ __host__ BitRef BitArray::operator[](size_t index) {
        return BitRef{ &data[index / 8], static_cast<size_t>(index % 8) };
    }
    __device__ __host__ uint8_t* BitArray::raw() { return data; }
    __device__ __host__ size_t BitArray::bit_size() const { return size; }
    __device__ __host__ size_t BitArray::byte_size() const {
		return (size + 7) / 8;
    }
    std::ostream& operator<<(std::ostream& os, const BitArray& bits) {
        for (size_t i = 0; i < bits.bit_size(); ++i) {
            os << bits[i];  // Print each bit
        }
        return os;
    }

    //2D
    __device__ bool ray_intersects_aabb(float2 start, float2 direction, float2 bmin, float2 bmax, float2* out_intersect, float2* out_normal) {
        float inv_dir_x = 1.0f / (direction.x == 0 ? FLT_EPS : direction.x);
        float inv_dir_y = 1.0f / (direction.y == 0 ? FLT_EPS : direction.y);

        float t_min_x = (bmin.x - start.x) * inv_dir_x;
        float t_max_x = (bmax.x - start.x) * inv_dir_x;
        float t_min_y = (bmin.y - start.y) * inv_dir_y;
        float t_max_y = (bmax.y - start.y) * inv_dir_y;

        float t1_x = fminf(t_min_x, t_max_x);
        float t2_x = fmaxf(t_min_x, t_max_x);
        float t1_y = fminf(t_min_y, t_max_y);
        float t2_y = fmaxf(t_min_y, t_max_y);

        float t_min = fmaxf(t1_x, t1_y);  // Largest entering time
        float t_max = fminf(t2_x, t2_y);  // Smallest exiting time

        if (t_max < fmaxf(t_min, 0.0f)) {
            return false;  // No intersection
        }
        if (out_intersect) {
            *out_intersect = make_float2(start.x + t_min * direction.x, start.y + t_min * direction.y);
        }
        if (out_normal) {
            // Determine the axis the intersection happened on
            if (t1_x > t1_y) {
                *out_normal = make_float2((inv_dir_x < 0.0f) ? -1.0f : 1.0f, 0.0f);
            }
            else {
                *out_normal = make_float2(0.0f, (inv_dir_y < 0.0f) ? -1.0f : 1.0f);
            }
        }
        return true;
    }

	///////// 2D /////////
    __device__ void dda_ray_traversal(
        DDARayParams<float2, 2> Params,
		DDARayResults<float2>& Results
    ) {
        float x = Params.start.x;
        float y = Params.start.y;
        float dx = Params.direction.x;
        float dy = Params.direction.y;

        int cell_x = static_cast<int>(x);
        int cell_y = static_cast<int>(y);

        int step_x = (dx > 0) ? 1 : -1;
        int step_y = (dy > 0) ? 1 : -1;

        float tDelta_x = (dx != 0) ? fabs(1.0f / dx) : FLT_INF;
        float tDelta_y = (dy != 0) ? fabs(1.0f / dy) : FLT_INF;

        float tMax_x = (dx != 0) ? (((cell_x + (step_x > 0)) - x) / dx) : FLT_INF;
        float tMax_y = (dy != 0) ? (((cell_y + (step_y > 0)) - y) / dy) : FLT_INF;

        Results.HitIntersectedPoint = make_float2(x, y);
        Results.hit = false;
        Results.isOutOfBounds = false;
        Results.stepsTaken = 0;

        int rows = Params.VoxelBuffer.dimensions[1];
        int cols = Params.VoxelBuffer.dimensions[0];
        auto grid = Params.VoxelBuffer.grid;

        for (int step = 0; step < Params.max_steps; ++step) {
            if (0 <= cell_x && cell_x < cols && 0 <= cell_y && cell_y < rows) {
                Results.HitCell = make_float2(cell_x, cell_y);
                int idx = (cell_y * cols + cell_x);
                if (Params.per_voxel_bounds) {
                    float bmin_x = Params.per_voxel_bounds[idx].min.x + cell_x * Params.per_voxel_bounds_scale;
                    float bmin_y = Params.per_voxel_bounds[idx].min.y + cell_y * Params.per_voxel_bounds_scale;
                    float bmax_x = Params.per_voxel_bounds[idx].max.x + 1 + cell_x * Params.per_voxel_bounds_scale;
                    float bmax_y = Params.per_voxel_bounds[idx].max.y + 1 + cell_y * Params.per_voxel_bounds_scale;
                    if (grid[idx] == 1 && bmin_x <= bmax_x) {
                        float temp_x = Params.start.x * Params.per_voxel_bounds_scale;
                        float temp_y = Params.start.y * Params.per_voxel_bounds_scale;
                        float2 aabb_normal = make_float2(0, 0);
                        if (GPUDDA::ray_intersects_aabb(make_float2(temp_x, temp_y), Params.direction, make_float2(bmin_x, bmin_y), make_float2(bmax_x, bmax_y), nullptr, &aabb_normal)) {
                            Results.hit = true;
                            if (step == 0) {
                                Results.HitNormal = aabb_normal;
                            }
                            break;
                        }
                    }
                }
                else {
                    if (grid[idx] == 1) {
                        Results.hit = true;
                        break;
                    }
                }
            }
            else {
                Results.isOutOfBounds = true;
                break;
            }

            float intersect_x = 0;
            float intersect_y = 0;
            if (tMax_x < tMax_y) {
                intersect_x = cell_x + (step_x > 0);
                intersect_y = y + (tMax_x * dy);
                cell_x += step_x;
                tMax_x += tDelta_x;
                Results.HitNormal = make_float2(step_x, 0);
            }
            else {
                intersect_x = x + (tMax_y * dx);
                intersect_y = cell_y + (step_y > 0);
                cell_y += step_y;
                tMax_y += tDelta_y;
                Results.HitNormal = make_float2(0, step_y);
            }

            if (Params.bounds) {
                int min_x = Params.bounds->min.x;
                int min_y = Params.bounds->min.y;
                int max_x = Params.bounds->max.x;
                int max_y = Params.bounds->max.y;
				bool isOutOfBounds = (intersect_x < min_x || intersect_x > max_x || intersect_y < min_y || intersect_y > max_y);
                if (isOutOfBounds) {
                    Results.isOutOfBounds = true;
                    break;
                }
            }

            Results.stepsTaken += 1;
            Results.HitIntersectedPoint = make_float2(intersect_x, intersect_y);
        }
    }

    __device__ float2 raytrace(float2 origin, float2 ray, VoxelBuffer<2> chunks, VoxelBuffer<2>* chunksData, Bounds<float2>* chunkBoundingBoxes, int factor,
        int& out_steps, float2& out_normal) {
        float rayLen = sqrt(ray.x * ray.x + ray.y * ray.y);
        ray.x /= rayLen;
        ray.y /= rayLen;

        float2 previous_cell = make_float2(-1, -1);
        int total_steps = 0;

        //in chunk space
        float2 start = origin;
        start.x /= factor;
        start.y /= factor;
        float2 direction = normalize(ray);
        float eps = FLT_EPS_DDA;
        if (!(start.x >= 0 && start.y >= 0 && start.x < chunks.dimensions[0] && start.y < chunks.dimensions[1])) {
            float2 intersect;
            if (GPUDDA::ray_intersects_aabb(start, direction, make_float2(0, 0), make_float2(chunks.dimensions[0], chunks.dimensions[1]), &intersect, nullptr)) {
                if (intersect.x == chunks.dimensions[0])
                    intersect.x -= 1;
                if (intersect.y == chunks.dimensions[1])
                    intersect.y -= 1;
                start = intersect;
            }
        }
		out_normal = make_float2(0, 0);
        float2 hitPosition = make_float2(0, 0);
        bool hit = false;
        while (true) {

            float2 start_high_res;
            DDARayParams<float2, 2> params = DDARayParams<float2, 2>::Default(chunks, start, direction);
            params.per_voxel_bounds = chunkBoundingBoxes;
            params.per_voxel_bounds_scale = factor;
            DDARayResults<float2> results;
            dda_ray_traversal(params, results);

            total_steps += results.stepsTaken;
            start_high_res = make_float2(results.HitIntersectedPoint.x * factor, results.HitIntersectedPoint.y * factor);
            hitPosition = start_high_res;

            if (results.hit && !results.isOutOfBounds) {
                float4 chunkBounds{};
                if (previous_cell.x == results.HitCell.x && previous_cell.y == results.HitCell.y) {
                    break;
                }
                previous_cell = results.HitCell;
                chunkBounds.x = 0;
                chunkBounds.y = 0;
                chunkBounds.z = factor;
                chunkBounds.w = factor;

                start_high_res.x -= results.HitCell.x * factor;
                start_high_res.y -= results.HitCell.y * factor;
                VoxelBuffer<2> chunkData = chunksData[(int)(results.HitCell.y * chunks.dimensions[0] + results.HitCell.x)];

                if (start_high_res.x == factor)
                    start_high_res.x -= eps;
                if (start_high_res.y == factor)
                    start_high_res.y -= eps;

                DDARayParams<float2, 2> params_hr = DDARayParams<float2, 2>::Default(chunkData, start_high_res, direction);
                params_hr.bounds = reinterpret_cast<Bounds<float2>*>(&chunkBounds);
                DDARayResults<float2> results_hr;

                dda_ray_traversal(params_hr, results_hr);

                total_steps += results_hr.stepsTaken;

                hitPosition = results_hr.HitIntersectedPoint;
                hitPosition.x += results.HitCell.x * factor;
                hitPosition.y += results.HitCell.y * factor;

                if (!results_hr.hit) {

                    start = make_float2(
                        results_hr.HitIntersectedPoint.x + results.HitCell.x * factor,
                        results_hr.HitIntersectedPoint.y + results.HitCell.y * factor
                    );
                    start.x /= factor;
                    start.y /= factor;

                    if (direction.x < 0)
                        start.x -= eps;
                    if (direction.y < 0)
                        start.y -= eps;

                    continue;
                }
                else {
                    if (results_hr.stepsTaken == 0) {
                        out_normal = results.HitNormal;
                    }
                    else {
                        out_normal = results_hr.HitNormal;
                    }
					hit = true;
                    break;
                }
            }
            else {
                break;
            }
        }
		out_steps = total_steps;
        return hit ? hitPosition : make_float2(FLT_INF, FLT_INF);
    }

	__global__ void dispatch(
		float2* origins,
		float2* rays,
		VoxelBuffer<2>* chunks,
		VoxelBuffer<2>* chunksData,
		Bounds<float2>* chunkBoundingBoxes,
		int factor,
		float2* results_point,
		float2* results_normal,
		int* results_steps,
		int count
	) {
		int idx = blockIdx.x * blockDim.x + threadIdx.x;
		if (idx < count) {
            int steps;
			float2 normal;
            results_point[idx] = raytrace(origins[idx], rays[idx], chunks[0], chunksData, chunkBoundingBoxes, factor, steps, normal);
			results_steps[idx] = steps;
			results_normal[idx] = normal;
		}
	}


    void VoxelRaytracer2D::UploadVoxelBuffer(const GPUDDA::VoxelBuffer<2>& buff) {
        if (gpu_VoxelBuffer == nullptr) {
            hipMalloc((void**)&gpu_VoxelBuffer, sizeof(GPUDDA::VoxelBuffer<2>));
            gpu_VoxelBufferGrid = BitArray(buff.grid.bit_size(), true);
        }

        dimensions.x = buff.dimensions[0];
		dimensions.y = buff.dimensions[1];

        VoxelBuffer<2> temp;
        temp.dimensions[0] = buff.dimensions[0];
        temp.dimensions[1] = buff.dimensions[1];
        gpu_VoxelBufferGrid.Copy(buff.grid, true);
        temp.grid = gpu_VoxelBufferGrid;
        hipMemcpy(gpu_VoxelBuffer, &temp, sizeof(GPUDDA::VoxelBuffer<2>), hipMemcpyHostToDevice);
    }

    void VoxelRaytracer2D::UploadVoxelBufferDatas(GPUDDA::VoxelBuffer<2>* buff, size_t count) {
        auto memSize = sizeof(GPUDDA::VoxelBuffer<2>) * count;
        if (gpu_VoxelBufferDatas == nullptr) {
            hipMalloc((void**)&gpu_VoxelBufferDatas, memSize);
            for (size_t i = 0; i < count; i++)
            {
                gpu_VoxelBufferDatasGrid.push_back(BitArray(buff[i].grid.bit_size(), true));
            }
        }

        GPUDDA::VoxelBuffer<2>* temp = new GPUDDA::VoxelBuffer<2>[count];
        for (size_t i = 0; i < count; i++)
        {
            temp[i].dimensions[0] = buff[i].dimensions[0];
            temp[i].dimensions[1] = buff[i].dimensions[1];
            gpu_VoxelBufferDatasGrid[i].Copy(buff[i].grid, true);
            temp[i].grid = gpu_VoxelBufferDatasGrid[i];
        }

        hipMemcpy(gpu_VoxelBufferDatas, temp, memSize, hipMemcpyHostToDevice);
    }

    void VoxelRaytracer2D::UploadVoxelBufferDataBounds(Bounds<float2>* bounds, size_t count) {
        auto memSize = sizeof(Bounds<float2>) * count;
        if (gpu_VoxelBufferDataBounds == nullptr) {
            hipMalloc((void**)&gpu_VoxelBufferDataBounds, memSize);
        }
        hipMemcpy(gpu_VoxelBufferDataBounds, bounds, memSize, hipMemcpyHostToDevice);
    }

    RayTraceResults<float2> VoxelRaytracer2D::Raytrace(std::vector<float2> origin, std::vector<float2> ray) {
		hipDeviceSynchronize();

		auto t0 = std::chrono::high_resolution_clock::now();

		auto result = resultsCPU;
        int count = origin.size();

        hipMemcpy(d_origins, origin.data(), sizeof(float2) * count, hipMemcpyHostToDevice);
        hipMemcpy(d_rays, ray.data(), sizeof(float2) * count, hipMemcpyHostToDevice);

        dim3 blockSize(8, 1, 1);
        dim3 numBlocks((count + (count - 1) / blockSize.x), 1, 1);

        dispatch << < numBlocks, blockSize >> > (
            d_origins, d_rays, gpu_VoxelBuffer, gpu_VoxelBufferDatas, gpu_VoxelBufferDataBounds, factor, 
            d_results, d_results_normal, d_results_steps, count);

		hipMemcpy(result.hitPoint.get(), d_results, sizeof(float2) * count, hipMemcpyDeviceToHost);
		hipMemcpy(result.normal.get(), d_results_normal, sizeof(float2) * count, hipMemcpyDeviceToHost);
		hipMemcpy(result.steps.get(), d_results_steps, sizeof(int) * count, hipMemcpyDeviceToHost);

		auto validPtr = result.valid.get();
		auto pointPtr = result.hitPoint.get();
		auto distancePtr = result.distance.get();
		auto voxelPtr = result.voxelIndex.get();
        for (size_t i = 0; i < count; i++)
        {
			validPtr[i] = (pointPtr[i].x != FLT_INF && pointPtr[i].y != FLT_INF);
            if (validPtr[i]) {
                float dtx = origin[i].x - pointPtr[i].x;
                float dty = origin[i].y - pointPtr[i].y;
				distancePtr[i] = sqrt(dtx * dtx + dty * dty);
				voxelPtr[i] = (int)(pointPtr[i].y * dimensions.x + pointPtr[i].x);
            }
        }

		hipDeviceSynchronize();

		auto t1 = std::chrono::high_resolution_clock::now();
		auto dt = std::chrono::duration_cast<std::chrono::microseconds>(t1 - t0).count();
		std::cout << "Raytracing time: " << dt / 1000.0f << " ms" << std::endl;

        return result;
    }

	///////// 3D /////////
	__global__ void dispatch(
		float3* origins,
		float3* rays,
		VoxelBuffer<3>* chunks,
		VoxelBuffer<3>* chunksData,
		Bounds<float3>* chunkBoundingBoxes,
		int factor,
		float3* results_point,
		float3* results_normal,
		int* results_steps,
		int count
	) {
		int idx = blockIdx.x * blockDim.x + threadIdx.x;
		if (idx < count) {
			int steps;
			float3 normal;
            float3 pos;
            if (raytrace(MAX_STEPS, origins[idx], rays[idx], chunks[0], chunksData, chunkBoundingBoxes, factor, steps, normal, pos)) {
				results_point[idx] = pos;
            }
            else {
				results_point[idx] = make_float3(FLT_INF, FLT_INF, FLT_INF);
            }
			results_steps[idx] = steps;
			results_normal[idx] = normal;
		}
	}

    __device__ bool ray_intersects_aabb(const float3& start, const float3 &direction, const float3 &bmin, const float3 &bmax, float3* out_intersect, float3* out_normal) {
        float inv_dir_x = 1.0f / (direction.x == 0 ? FLT_EPS : direction.x);
        float inv_dir_y = 1.0f / (direction.y == 0 ? FLT_EPS : direction.y);
        float inv_dir_z = 1.0f / (direction.z == 0 ? FLT_EPS : direction.z);

        float t_min_x = (bmin.x - start.x) * inv_dir_x;
        float t_max_x = (bmax.x - start.x) * inv_dir_x;
        float t_min_y = (bmin.y - start.y) * inv_dir_y;
        float t_max_y = (bmax.y - start.y) * inv_dir_y;
        float t_min_z = (bmin.z - start.z) * inv_dir_z;
        float t_max_z = (bmax.z - start.z) * inv_dir_z;

        float t1_x = fminf(t_min_x, t_max_x);
        float t2_x = fmaxf(t_min_x, t_max_x);
        float t1_y = fminf(t_min_y, t_max_y);
        float t2_y = fmaxf(t_min_y, t_max_y);
        float t1_z = fminf(t_min_z, t_max_z);
        float t2_z = fmaxf(t_min_z, t_max_z);

        float t_min = fmaxf(fmaxf(t1_x, t1_y), t1_z);  // Largest entering time
        float t_max = fminf(fminf(t2_x, t2_y), t2_z);  // Smallest exiting time

        if (t_max < fmaxf(t_min, 0.0f)) {
            return false;  // No intersection
        }
        if (out_intersect) {
            *out_intersect = make_float3(start.x + t_min * direction.x, start.y + t_min * direction.y, start.z + t_min * direction.z);
        }

        if (out_normal) {
            if (t_min == t1_x) {
                *out_normal = make_float3((inv_dir_x < 0.0f) ? -1.0f : 1.0f, 0.0f, 0.0f);
            }
            else if (t_min == t1_y) {
                *out_normal = make_float3(0.0f, (inv_dir_y < 0.0f) ? -1.0f : 1.0f, 0.0f);
            }
            else { // t_min == t1_z
                *out_normal = make_float3(0.0f, 0.0f, (inv_dir_z < 0.0f) ? -1.0f : 1.0f);
            }
        }

        return true;
    }

    __device__ void dda_ray_traversal(
        DDARayParams<float3, 3> Params,
        DDARayResults<float3>& Results
    ) {
        float x = Params.start.x;
        float y = Params.start.y;
        float z = Params.start.z;

        float dx = Params.direction.x;
        float dy = Params.direction.y;
        float dz = Params.direction.z;

        int cell_x = static_cast<int>(x);
        int cell_y = static_cast<int>(y);
        int cell_z = static_cast<int>(z);

        //start tracing
        int depth = Params.VoxelBuffer.dimensions[2];
        int rows = Params.VoxelBuffer.dimensions[1];
        int cols = Params.VoxelBuffer.dimensions[0];

        int step_x = (dx > 0) ? 1 : -1;
        int step_y = (dy > 0) ? 1 : -1;
        int step_z = (dz > 0) ? 1 : -1;

        float tDelta_x = (dx != 0) ? fabs(1.0f / dx) : FLT_INF;
        float tDelta_y = (dy != 0) ? fabs(1.0f / dy) : FLT_INF;
        float tDelta_z = (dz != 0) ? fabs(1.0f / dz) : FLT_INF;

        float tMax_x = (dx != 0) ? (((cell_x + (step_x > 0)) - x) / dx) : FLT_INF;
        float tMax_y = (dy != 0) ? (((cell_y + (step_y > 0)) - y) / dy) : FLT_INF;
        float tMax_z = (dz != 0) ? (((cell_z + (step_z > 0)) - z) / dz) : FLT_INF;

        DDARayResults<float3> tResults;

        tResults.HitIntersectedPoint = make_float3(x, y, z);
        tResults.hit = false;
        tResults.isOutOfBounds = false;
        tResults.stepsTaken = 0;

        auto grid = Params.VoxelBuffer.grid;
        bool exit = false;

		bool IsOnEdge = cell_x == cols || cell_y == rows || cell_z == depth;
        float3 edgePadding = make_float3(0,0,0);
        if (IsOnEdge) {
            if (dx < 0) {
                edgePadding.x = 1;
            }
			if (dy < 0) {
				edgePadding.y = 1;
			}
			if (dz < 0) {
				edgePadding.z = 1;
			}
        }
        int maxSteps = Params.max_steps;
        for (int step = 0; step < maxSteps; ++step) {
            if (0 <= cell_x && cell_x < cols + edgePadding.x &&
                0 <= cell_y && cell_y < rows + edgePadding.y &&
                0 <= cell_z && cell_z < depth + edgePadding.z) {

                int clamped_x = min(max(cell_x, 0), cols - 1);
                int clamped_y = min(max(cell_y, 0), rows - 1);
                int clamped_z = min(max(cell_z, 0), depth - 1);
                tResults.HitCell = make_float3(clamped_x, clamped_y, clamped_z);
                int idx = (clamped_z * rows * cols + clamped_y * cols + clamped_x);
                bool isVoxel = grid[idx] == 1;
                if (Params.per_voxel_bounds) {
                    auto bounds = Params.per_voxel_bounds[idx];
                    float bmin_x = bounds.min.x + clamped_x * Params.per_voxel_bounds_scale;
                    float bmin_y = bounds.min.y + clamped_y * Params.per_voxel_bounds_scale;
                    float bmin_z = bounds.min.z + clamped_z * Params.per_voxel_bounds_scale;
                    float bmax_x = bounds.max.x + 1 + clamped_x * Params.per_voxel_bounds_scale;
                    float bmax_y = bounds.max.y + 1 + clamped_y * Params.per_voxel_bounds_scale;
                    float bmax_z = bounds.max.z + 1 + clamped_z * Params.per_voxel_bounds_scale;
                    if (isVoxel && bmin_x <= bmax_x) {
                        float temp_x = Params.start.x * Params.per_voxel_bounds_scale;
                        float temp_y = Params.start.y * Params.per_voxel_bounds_scale;
                        float temp_z = Params.start.z * Params.per_voxel_bounds_scale;
                        float3 aabb_normal = make_float3(0, 0, 0);
                        if (ray_intersects_aabb(make_float3(temp_x, temp_y, temp_z), Params.direction,
                            make_float3(bmin_x, bmin_y, bmin_z), make_float3(bmax_x, bmax_y, bmax_z), nullptr, &aabb_normal)) {
                            tResults.hit = true;
                            if (step == 0) {
                                tResults.HitNormal = aabb_normal;
                            }
                            exit = true;
                        }
                    }
                }
                else {
                    if (isVoxel) {
                        tResults.hit = true;
                        exit = true;
                    }
                }
            }
            else {
                tResults.isOutOfBounds = true;
                exit = true;
            }

            float intersect_x = 0;
            float intersect_y = 0;
            float intersect_z = 0;
            if (tMax_x < tMax_y && tMax_x < tMax_z) {
                intersect_x = cell_x + (step_x > 0);
                intersect_y = y + (tMax_x * dy);
                intersect_z = z + (tMax_x * dz);
                cell_x += step_x;
                tMax_x += tDelta_x;
                if (!exit) 
                    tResults.HitNormal = make_float3(step_x, 0, 0);
            }
            else if (tMax_y <= tMax_x && tMax_y < tMax_z) {
                intersect_x = x + (tMax_y * dx);
                intersect_y = cell_y + (step_y > 0);
                intersect_z = z + (tMax_y * dz);
                cell_y += step_y;
                tMax_y += tDelta_y;
                if (!exit) 
                    tResults.HitNormal = make_float3(0, step_y, 0);
            }
            else {
                intersect_x = x + (tMax_z * dx);
                intersect_y = y + (tMax_z * dy);
                intersect_z = cell_z + (step_z > 0);
                cell_z += step_z;
                tMax_z += tDelta_z;
                if (!exit) 
                    tResults.HitNormal = make_float3(0, 0, step_z);
            }
            if (!exit) {
                if (Params.bounds) {
                    int min_x = Params.bounds->min.x;
                    int min_y = Params.bounds->min.y;
                    int min_z = Params.bounds->min.z;
                    int max_x = Params.bounds->max.x;
                    int max_y = Params.bounds->max.y;
                    int max_z = Params.bounds->max.z;
                    // Check if the intersection point is within the bounds
                    bool isOutOfBounds = (intersect_x < min_x || intersect_x > max_x || intersect_y < min_y || intersect_y > max_y || intersect_z < min_z || intersect_z > max_z);
                    if (isOutOfBounds) {
                        tResults.isOutOfBounds = true;
                        break;
                    }
                }
                tResults.stepsTaken += 1;
                tResults.HitIntersectedPoint = make_float3(intersect_x, intersect_y, intersect_z);
            }
            else {
                tResults.NextCell = make_float3(cell_x, cell_y, cell_z);
                tResults.NextInterSectedPoint = make_float3(intersect_x, intersect_y, intersect_z);
                break;
            }
        }
        Results = tResults;
    }
    __device__ bool raytrace(int maxSteps, float3 origin, float3 ray, VoxelBuffer<3> chunks, VoxelBuffer<3>* chunksData, Bounds<float3>* chunkBoundingBoxes, int factor,
        int& out_steps, float3& out_normal, float3& out_pos) {
        
        float rayLen = sqrt(ray.x * ray.x + ray.y * ray.y + ray.z * ray.z);
        ray.x /= rayLen;
        ray.y /= rayLen;
        ray.z /= rayLen;
        float3 previous_cell = make_float3(-1, -1, -1);
        int total_steps = 0;
        //in chunk space
        float invFactor = 1.0f / factor;
        float3 start = origin;
        start.x *= invFactor;
        start.y *= invFactor;
        start.z *= invFactor;
        float3 direction = normalize(ray);
		float3 start_normal = make_float3(0, 0, 0);
        if (!(start.x >= 0 && start.y >= 0 && start.z >= 0 && start.x < chunks.dimensions[0] && start.y < chunks.dimensions[1] && start.z < chunks.dimensions[2])) {
            float3 intersect;
            if (ray_intersects_aabb(
                make_float3(start.x, start.y, start.z), 
                direction, 
                make_float3(FLT_EPS_DDA, FLT_EPS_DDA, FLT_EPS_DDA),
                make_float3(chunks.dimensions[0] - FLT_EPS_DDA, chunks.dimensions[1] - FLT_EPS_DDA, chunks.dimensions[2] - FLT_EPS_DDA),
                &intersect,
                &start_normal)) {
                start = intersect;
            }
            else {
                return false;
            }
        }
		out_normal = make_float3(0, 0, 0);
        float3 hitPosition = make_float3(0, 0, 0);
        bool hit = false;

        while (total_steps < maxSteps) {
            float3 start_high_res;
            DDARayParams<float3, 3> params = DDARayParams<float3, 3>::Default(chunks, start, direction);
            params.per_voxel_bounds = chunkBoundingBoxes;
            params.per_voxel_bounds_scale = factor;
            DDARayResults<float3> results;
            dda_ray_traversal(params, results);

            total_steps += results.stepsTaken;
            start_high_res = make_float3(results.HitIntersectedPoint.x * factor, results.HitIntersectedPoint.y * factor, results.HitIntersectedPoint.z * factor);
            hitPosition = start_high_res;
            if (results.hit && !results.isOutOfBounds) {
                Bounds<float3> chunkBounds{};
                if (previous_cell.x == results.HitCell.x && 
                    previous_cell.y == results.HitCell.y && 
                    previous_cell.z == results.HitCell.z) {
                    break;
                }
                previous_cell = results.HitCell;
                chunkBounds.min.x = 0;
                chunkBounds.min.y = 0;
                chunkBounds.min.z = 0;
                chunkBounds.max.x = factor;
                chunkBounds.max.y = factor;
                chunkBounds.max.z = factor;
                start_high_res.x -= results.HitCell.x * factor;
                start_high_res.y -= results.HitCell.y * factor;
                start_high_res.z -= results.HitCell.z * factor;

                VoxelBuffer<3> chunkData = chunksData[(int)(results.HitCell.z * chunks.dimensions[1] * chunks.dimensions[0] + results.HitCell.y * chunks.dimensions[0] + results.HitCell.x)];
                DDARayParams<float3, 3> params_hr = DDARayParams<float3, 3>::Default(chunkData, start_high_res, direction);
                params_hr.bounds = &chunkBounds;
                DDARayResults<float3> results_hr;
                dda_ray_traversal(params_hr, results_hr);

                total_steps += results_hr.stepsTaken;
                hitPosition = make_float3(
                    results_hr.HitIntersectedPoint.x + results.HitCell.x * factor,
                    results_hr.HitIntersectedPoint.y + results.HitCell.y * factor,
                    results_hr.HitIntersectedPoint.z + results.HitCell.z * factor
                );
                
                if (!results_hr.hit) {
                    start = hitPosition;
                    start *= invFactor;

                    bool firstPass = false;

                    //push towards next cell if voxel dda ended on chunk border
                    if (results_hr.isOutOfBounds) {
                        float cx = floorf(start.x);
                        float cy = floorf(start.y);
                        float cz = floorf(start.z);
                        if (results.HitCell.x == cx) {
                            firstPass = true;
                            start.x = direction.x < 0 ? nextafterf(start.x, -FLT_INF) : start.x;
                        }
                        if (results.HitCell.y == cy) {
                            firstPass = true;
                            start.y = direction.y < 0 ? nextafterf(start.y, -FLT_INF) : start.y;
                        }
                        if (results.HitCell.z == cz) {
                            firstPass = true;
                            start.z = direction.z < 0 ? nextafterf(start.z, -FLT_INF) : start.z;
                        }
                    }

                    //if still in same cell, find smallest diff to next cell
                    if (firstPass) {
                        float cx = floorf(start.x);
                        float cy = floorf(start.y);
                        float cz = floorf(start.z);
                        if (cx == results.HitCell.x && cy == results.HitCell.y && cz == results.HitCell.z) {
                            //find smallest diff to next cell
                            float3 diff = make_float3(
                                results.NextCell.x - start.x,
                                results.NextCell.y - start.y,
                                results.NextCell.z - start.z
                            );

                            float3 absDiff = make_float3(
                                fabsf(diff.x),
                                fabsf(diff.y),
                                fabsf(diff.z)
                            );

                            if (absDiff.x < absDiff.y && absDiff.x < absDiff.z) {
                                start.x += diff.x;
                            }
                            else if (absDiff.y < absDiff.x && absDiff.y < absDiff.z) {
                                start.y += diff.y;
                            }
                            else {
                                start.z += diff.z;
                            }
                        }
                    }

                    continue;
                }
                else {
                    //steps taken was 0, use chunk's normal
					if (results_hr.stepsTaken == 0) {
						out_normal = results.HitNormal;
					}
					else {
						out_normal = results_hr.HitNormal;
					}
                    hit = true;
                    break;
                }
            }
            else {
                break;
            }
        }
        out_steps = total_steps;
        if (hit) {
            out_pos = hitPosition;
            if (total_steps == 0) {
                out_pos = start * factor;
                out_normal = start_normal;
            }
        }
        return hit;
    }

    void VoxelRaytracer3D::UploadVoxelBuffer(const GPUDDA::VoxelBuffer<3>& buff) {
        if (gpu_VoxelBuffer == nullptr) {
            hipMalloc((void**)&gpu_VoxelBuffer, sizeof(GPUDDA::VoxelBuffer<3>));
            gpu_VoxelBufferGrid = BitArray(buff.grid.bit_size(), true);
        }

		dimensions.x = buff.dimensions[0];
		dimensions.y = buff.dimensions[1];
		dimensions.z = buff.dimensions[2];
		VoxelBuffer<3> temp;
        gpu_VoxelBufferGrid.Copy(buff.grid, true);

        temp.grid = gpu_VoxelBufferGrid;
		temp.dimensions[0] = buff.dimensions[0];
		temp.dimensions[1] = buff.dimensions[1];
		temp.dimensions[2] = buff.dimensions[2];
		hipMemcpy(gpu_VoxelBuffer, &temp, sizeof(GPUDDA::VoxelBuffer<3>), hipMemcpyHostToDevice);
	}

    void AsyncFuncThread(VoxelRaytracer3D* raytracer) {
        VoxelRaytracer3D* t = raytracer;
    	while (t->doAsyncThread) {
            //wait for new params
            while (true) {
                if (!t->doAsyncThread) {
                    return;
                }

                auto value = t->async_params_loaded.load(std::memory_order_acquire);
                if (value == false) {
                    continue;
                }
                else {
                    break;
                }
            }

            auto params = *t->async_params;

            //chunks
            t->gpu_VoxelBufferGrid.AsyncCopy(params.chunks->grid, t->stream);
            //voxels
            for (size_t i = 0; i < params.voxelsCount; i++)
            {
                t->gpu_VoxelBufferDatasGrid[i].AsyncCopy(params.voxels[i].grid, t->stream);
            }
            //bounds
            auto memSize = sizeof(Bounds<float3>) * params.boundsCount;
            hipMemcpyAsync(t->gpu_VoxelBufferDataBounds, params.bounds, memSize, hipMemcpyHostToDevice, t->stream);

            //wait for all copies to finish
            hipStreamSynchronize(t->stream);

            t->dimensions.x = params.chunks->dimensions[0];
            t->dimensions.y = params.chunks->dimensions[1];
            t->dimensions.z = params.chunks->dimensions[2];

            //chunks final
            VoxelBuffer<3> chunks;
            chunks.grid = t->gpu_VoxelBufferGrid;
            chunks.dimensions[0] = params.chunks->dimensions[0];
            chunks.dimensions[1] = params.chunks->dimensions[1];
            chunks.dimensions[2] = params.chunks->dimensions[2];
            hipMemcpyAsync(t->gpu_VoxelBuffer, &chunks, sizeof(GPUDDA::VoxelBuffer<3>), hipMemcpyHostToDevice, t->stream);

            //voxels final
            std::vector<GPUDDA::VoxelBuffer<3>> temp(params.voxelsCount);
            memSize = sizeof(GPUDDA::VoxelBuffer<3>) * params.voxelsCount;
            for (size_t i = 0; i < params.voxelsCount; i++)
            {
                temp[i].dimensions[0] = params.voxels[i].dimensions[0];
                temp[i].dimensions[1] = params.voxels[i].dimensions[1];
                temp[i].dimensions[2] = params.voxels[i].dimensions[2];
                temp[i].grid = t->gpu_VoxelBufferDatasGrid[i];
            }
            hipMemcpyAsync(t->gpu_VoxelBufferDatas, temp.data(), memSize, hipMemcpyHostToDevice, t->stream);

            //wait for all copies to finish
            hipStreamSynchronize(t->stream);

            delete t->async_params;
            t->async_params = nullptr;
            t->async_params_loaded.store(false, std::memory_order_release);
            t->AsyncOperationState = 2;
        }
    }
    VoxelRaytracer3D::~VoxelRaytracer3D() {
        doAsyncThread = false;
        if (async_thread.size() > 0) {
            if (async_thread[0].joinable()) {
                async_thread[0].join();
            }
		}
        async_thread.clear();
    }
    bool VoxelRaytracer3D::UploadBuffersAsync(
        const GPUDDA::VoxelBuffer<3>& chunks,
        GPUDDA::VoxelBuffer<3>* voxels, size_t voxelsCount,
        Bounds<float3>* bounds, size_t boundsCount
        ) 
    {
        //if not in idle state, do not start another upload
        if(AsyncOperationState != 0) {
            throw std::runtime_error("Async upload already in progress");
		}

        doAsyncThread = true;

        if (async_thread.size() == 0) {
            async_thread.push_back(std::thread(AsyncFuncThread, this));
        }

        //chunks
        if (gpu_VoxelBuffer == nullptr) {
            hipMalloc((void**)&gpu_VoxelBuffer, sizeof(GPUDDA::VoxelBuffer<3>));
            gpu_VoxelBufferGrid = BitArray(chunks.grid.bit_size(), true);
        }
        //voxels
        auto memSize = sizeof(GPUDDA::VoxelBuffer<3>) * voxelsCount;
        if (gpu_VoxelBufferDatas == nullptr) {
            hipMalloc((void**)&gpu_VoxelBufferDatas, memSize);
            for (size_t i = 0; i < voxelsCount; i++)
            {
                gpu_VoxelBufferDatasGrid.push_back(BitArray(voxels[i].grid.bit_size(), true));
            }
        }
        //bounds
        memSize = sizeof(Bounds<float3>) * boundsCount;
        if (gpu_VoxelBufferDataBounds == nullptr) {
            hipMalloc((void**)&gpu_VoxelBufferDataBounds, memSize);
        }

        AsyncOperationState = 1;
        AsyncParams* params = new AsyncParams();
        params->chunks = &chunks;
        params->voxels = voxels;
        params->voxelsCount = voxelsCount;
        params->bounds = bounds;
        params->boundsCount = boundsCount;

        //send data to thread
        async_params = params;
        async_params_loaded.store(true, std::memory_order_release);

        return true;
    }
    void VoxelRaytracer3D::FinishAsyncUpload() {
        if (AsyncOperationState == 2) {
            AsyncOperationState = 0;
        }
        else {
            throw std::runtime_error("Async upload not finished yet");
        }
    }
    int VoxelRaytracer3D::IsUploadVoxelBufferAsyncComplete() {
        return AsyncOperationState;
	}

	void VoxelRaytracer3D::UploadVoxelBufferDatas(GPUDDA::VoxelBuffer<3>* buff, size_t count) {
        auto memSize = sizeof(GPUDDA::VoxelBuffer<3>) * count;
        if (gpu_VoxelBufferDatas == nullptr) {
            hipMalloc((void**)&gpu_VoxelBufferDatas, memSize);
            for (size_t i = 0; i < count; i++)
            {
                gpu_VoxelBufferDatasGrid.push_back(BitArray(buff[i].grid.bit_size(), true));
            }
        }

		GPUDDA::VoxelBuffer<3>* temp = new GPUDDA::VoxelBuffer<3>[count];
		for (size_t i = 0; i < count; i++)
		{
			temp[i].dimensions[0] = buff[i].dimensions[0];
			temp[i].dimensions[1] = buff[i].dimensions[1];
			temp[i].dimensions[2] = buff[i].dimensions[2];
            gpu_VoxelBufferDatasGrid[i].Copy(buff[i].grid, true);
            temp[i].grid = gpu_VoxelBufferDatasGrid[i];
		}
		hipMemcpy(gpu_VoxelBufferDatas, temp, memSize, hipMemcpyHostToDevice);
        delete[] temp;
	}

	void VoxelRaytracer3D::UploadVoxelBufferDataBounds(Bounds<float3>* bounds, size_t count) {
        auto memSize = sizeof(Bounds<float3>) * count;
        if (gpu_VoxelBufferDataBounds == nullptr) {
            hipMalloc((void**)&gpu_VoxelBufferDataBounds, memSize);
        }

		hipMemcpy(gpu_VoxelBufferDataBounds, bounds, memSize, hipMemcpyHostToDevice);
	}

    RayTraceResults<float3> VoxelRaytracer3D::Raytrace(std::vector<float3> origin, std::vector<float3> ray) {
		auto result = resultsCPU;
		int count = origin.size();

		hipMemcpy(d_origins, origin.data(), sizeof(float3) * count, hipMemcpyHostToDevice);
		hipMemcpy(d_rays, ray.data(), sizeof(float3) * count, hipMemcpyHostToDevice);

		dim3 blockSize(8, 1, 1);
		dim3 numBlocks((count + (count - 1) / blockSize.x), 1, 1);

		hipDeviceSynchronize();

		auto t0 = std::chrono::high_resolution_clock::now();
		dispatch <<< numBlocks, blockSize >> > (
			d_origins, d_rays, gpu_VoxelBuffer, gpu_VoxelBufferDatas, gpu_VoxelBufferDataBounds, factor,
			d_results, d_results_normal, d_results_steps, count);

		hipDeviceSynchronize();
		auto t1 = std::chrono::high_resolution_clock::now();
		auto dt = std::chrono::duration_cast<std::chrono::microseconds>(t1 - t0).count();
		std::cout << "Raytracing time: " << dt / 1000.0f << " ms" << std::endl;

		hipMemcpy(result.hitPoint.get(), d_results, sizeof(float3) * count, hipMemcpyDeviceToHost);
		hipMemcpy(result.normal.get(), d_results_normal, sizeof(float3) * count, hipMemcpyDeviceToHost);
		hipMemcpy(result.steps.get(), d_results_steps, sizeof(int) * count, hipMemcpyDeviceToHost);

		for (size_t i = 0; i < count; i++)
		{
			result.valid[i] = (result.hitPoint[i].x != FLT_INF && result.hitPoint[i].y != FLT_INF && result.hitPoint[i].z != FLT_INF);
			if (result.valid) {
				float dtx = origin[i].x - result.hitPoint[i].x;
				float dty = origin[i].y - result.hitPoint[i].y;
				float dtz = origin[i].z - result.hitPoint[i].z;
				result.distance[i] = sqrt(dtx * dtx + dty * dty + dtz * dtz);
				result.voxelIndex[i] = (int)(result.hitPoint[i].z * dimensions.x * dimensions.y + result.hitPoint[i].y * dimensions.x + result.hitPoint[i].x);
			}
		}

		hipDeviceSynchronize();
		return result;
    }


}
